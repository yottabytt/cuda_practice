/*
Tiny snippet to check the possibility of reading two consecutive float values in memory using a single call using float2
*/
#include <hip/hip_runtime.h>
#include <stdio.h>
__shared__ float a[32];
__global__ void sm_float2_set(){
    int tid = threadIdx.x;
    if(tid % 2 == 0){
        a[tid] = 15;
    }
    else{
        a[tid] = 7;
    }
    printf("%d set value %f", tid, a[tid] );
    __syncthreads();
}

__global__ void sm_float2_get(){
    float2 val = *(float2*)(a+threadIdx.x*2);
    printf("%d got value %f", threadIdx.x, val );
    printf("%d has two vals %f and %f", threadIdx.x, val.x, val.y);
}

int main(int argc, char **argv){
    sm_float2_set<<< 1,32 >>>();
    sm_float2_get<<< 1,16 >>>();
    hipDeviceReset();
    return 0;
}
